#include "hip/hip_runtime.h"
#include "unitTests.h"

template<typename T>
struct GForceTest : public testing::Test {
    using ParamType = T;
};

bool use_offline = true;

std::default_random_engine generator(0xffa0);

void random_vector(std::vector<double> &v, int size) {

    std::normal_distribution<double> distribution(0.0, 1.0);

    v.clear();
    v.resize(size);

    for (int i = 0; i < v.size(); i++) {
        v[i] = distribution(generator);
    }
}

TYPED_TEST_CASE(GForceTest, GFO<uint64_t>);

TYPED_TEST(GForceTest, SelectShare) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    Share x = {1, 2, 10, 1};
    Share y = {4, 5, 1, 6};
    Share b({1, 1, 0, 1}, false);

    Share z(x.size());
    selectShare(x, y, b, z);

    DeviceData<T> result(4);
    reconstruct(z, result);
    
    std::vector<double> expected = {4, 5, 10, 6};
    assertDeviceData(result, expected);
}

TYPED_TEST(GForceTest, SelectShare2) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    Share x = {1.3456, 2, 10, 1};
    Share y = {4.9999, 5, 1, 6.123456};
    Share b({1, 1, 0, 1}, false);

    Share z(x.size());
    selectShare(x, y, b, z);

    DeviceData<T> result(4);
    reconstruct(z, result);
    
    std::vector<double> expected = {4.9999, 5, 10, 6.123456};
    assertDeviceData(result, expected);
}

TYPED_TEST(GForceTest, Mult) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    Share a ({12, 24, 3, 5, -2, -3}, false); 
    Share b ({1, 0, 11, 3, -1, 11}, false);

    DeviceData<T> result(a.size());

    if (use_offline){
        b.offline_known = true;
    }
    a *= b;
    reconstruct(a, result);

    std::vector<double> expected = {12, 0, 33, 15, 2, -33};
    assertDeviceData(result, expected, false);
}

TYPED_TEST(GForceTest, MatMul) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    Share a = {1, 1, 1, 1, 1, 1};  // 2 x 3
    Share b = {1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1, 0}; // 3 x 4
    Share c(8); // 2 x 4

    DeviceData<T> result(8);

    if (use_offline){
        b.offline_known = true;
    }
    matmul(a, b, c, 2, 4, 3, false, false, false, (T)FLOAT_PRECISION);
    reconstruct(c, result);

    std::vector<double> expected = {1, 1, 1, 1, 0, 0, 1, 1};

    assertDeviceData(result, expected);
}

TYPED_TEST(GForceTest, MatMul2) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    Share a = {1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1};  // 4 x 4
    Share b = {-2.786461, -1.280988, -2.209210, 0.049379, 0.241369, 1.617007, -0.572261, 0.705014, -1.176370, -0.814461, 0.992866, 0.856274};
    Share c(12); // 4 x 3

    DeviceData<T> result(12);

    if (use_offline){
        b.offline_known = true;
    }
    matmul(a, b, c, 4, 3, 4, true, true, true, (T)FLOAT_PRECISION);
    reconstruct(c, result);

    std::vector<double> expected = {-2.786461, -1.280988, -2.209210, 0.049379, 0.241369, 1.617007, -0.572261, 0.705014, -1.176370, -0.814461, 0.992866, 0.856274};
    assertDeviceData(result, expected);
}

// TYPED_TEST(GForceTest, MULTIPLY_MAIN) {

//     using Share = typename TestFixture::ParamType;
//     using T = typename Share::share_type;

//     if (partyNum >= Share::numParties) return;

//     std::vector<double> vector_a, vector_b;

//     random_vector(vector_a, 20), random_vector(vector_b, 20);;
//     Share a(20), b(20);
//     a.setPublic(vector_a), b.setPublic(vector_b);

//     // DeviceData<T> test_data(20);
//     // test_data.fill(0);
//     // test_data += *a.getShare(0);
//     // test_data += GFORCE_BOUND;
//     // printDeviceData(test_data, "plus_bound", false);

//     if (use_offline){
//         b.offline_known = true;
//     }
//     a *= b;
//     dividePublic(a, (T)1 << FLOAT_PRECISION);
//     printDeviceData(*a.getShare(0), "actual", false);

//     std::vector<double> expected(20);
//     std::transform(vector_a.begin(), vector_a.end(), 
//         vector_b.begin(),  expected.begin(),
//         [](double x, double y) {return x*y;} );
//     DeviceData<T> expt(20);
//     thrust::copy(expected.begin(), expected.end(), expt.begin());

//     printDeviceData(expt, "expected", false);
//     assertShare(a, expected, false);
// }

TYPED_TEST(GForceTest, Convolution) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    size_t batchSize = 2;
    size_t imageWidth = 3, imageHeight = 3;
    size_t filterSize = 3;
    size_t Din = 1, Dout = 1;
    size_t stride = 1, padding = 1;

    // N=2, H=3, W=3, C=1
    Share im = {
        1, 2, 1, 2, 3, 2, 1, 2, 1,
        1, 2, 1, 2, 3, 2, 1, 2, 1
    };

    // N(Dout)=1, H=3, W=3, C(Din)=1
    Share filters = {
        1, 0, 1, 0, 1, 0, 1, 0, 1
    };

    size_t wKernels = (imageWidth - filterSize + (2*padding)) / stride + 1;
    size_t hKernels = (imageHeight - filterSize + (2*padding)) / stride + 1;
    Share out(batchSize * wKernels * hKernels * Dout);

    filters.offline_known = true;
    convolution(im, filters, out,
        cutlass::conv::Operator::kFprop,
        batchSize, imageHeight, imageWidth, filterSize,
        Din, Dout, stride, padding, FLOAT_PRECISION);

    std::vector<double> expected = {
        4, 6, 4, 6, 7, 6, 4, 6, 4,
        4, 6, 4, 6, 7, 6, 4, 6, 4,
    };
    DeviceData<T> result(out.size());
    reconstruct(out, result);
    assertDeviceData(result, expected);
}

TYPED_TEST(GForceTest, Truncate) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;
    // true
    printf("is GFO share?\t%d\n", typeid(Share)==typeid(GFO<T>));
    // false
    printf("is TPC share?\t%d\n", typeid(Share)==typeid(TPC<T>));

    Share a = {1 << 3, 2 << 3, 3 << 3, -3 << 3};
    dividePublic(a, (T)1 << 3);

    DeviceData<T> result(a.size());
    std::vector<double> expected = {1, 2, 3, -3};
    reconstruct(a, result);

    assertDeviceData(result, expected);
}

// TYPED_TEST(GForceTest, Truncate2) {

//     using Share = typename TestFixture::ParamType;
//     using T = typename Share::share_type;

//     if (partyNum >= Share::numParties) return;

//     Share a = {1 << 22};
//     dividePublic(a, (T)1 << 21);

//     DeviceData<T> result(a.size());
//     std::vector<double> expected = {2};
//     reconstruct(a, result);

//     assertDeviceData(result, expected);
// }

// TYPED_TEST(GForceTest, TruncateVec) {

//     using Share = typename TestFixture::ParamType;
//     using T = typename Share::share_type;

//     if (partyNum >= Share::numParties) return;

//     Share a = {1 << 3, 2 << 1, 3 << 0, -3 << 5, 2515014.0};

//     DeviceData<T> denominators(a.size());
//     denominators.fill(1);
//     DeviceData<T> pows = {3, 1, 0, 5, 2};
//     denominators <<= pows;

//     dividePublic(a, denominators);

//     DeviceData<T> result(a.size());
//     reconstruct(a, result);
//     std::vector<double> expected = {1, 2, 3, -3, 628753.5};
    
//     assertDeviceData(result, expected);
// }

TYPED_TEST(GForceTest, DRELU) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    Share input = {
        -1, -1, -1, -1, -1, -1, -1, -1,
        -1, -1, -1, -1, -1, -1, -1, -1,
        -1, -1, -1, -1, -1, -1, -1, -1,
        -1, -1, -1, -1, -1, -1, -1, -1,
        -1, 1
    };

    //Change Share to TPC<uint8_t>
    Share result(input.size());
    dReLU(input, result);

    std::vector<double> expected = {
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 1
    };

    //Change to <uint8_t>
    DeviceData<T> super_result(result.size());
    reconstruct(result, super_result);
    assertDeviceData(super_result, expected, false);
}

TYPED_TEST(GForceTest, DRELU2) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    Share input = {
        0.326922, 0.156987, 0.461417, -0.221444, 9.846086, 0.000000, 0.000000, 0.000000,
        0.326922, 0.156987, 0.461417, -0.221444, 9.846086, 0.000000, 0.000000, 0.000000,
        0.326922, 0.156987, 0.461417, -0.221444, 9.846086, 0.000000, 0.000000, 0.000000
    };

    //Change Share to TPC<uint8_t>
    Share result(input.size());
    dReLU(input, result);

    std::vector<double> expected = {
        1, 1, 1, 0, 1, 1, 1, 1,
        1, 1, 1, 0, 1, 1, 1, 1,
        1, 1, 1, 0, 1, 1, 1, 1
    };

    //Change to <uint8_t>
    DeviceData<T> super_result(result.size());
    reconstruct(result, super_result);

    printDeviceData(super_result, "actual", false);
    assertDeviceData(super_result, expected, false);
}

TYPED_TEST(GForceTest, DRELU3) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    Share input = {
        0.326922, 0.156987, 0.461417, -0.221444, 9.846086, 0.000000, 0.000000, 0.000000
    };

    T negative = (T)(-10 * (1 << FLOAT_PRECISION));
    DeviceData<T> add = {
        0, 0, 0, 0, 0, negative, negative, negative
    };
    for(int share = 0; share < Share::numShares(); share++) {
        *input.getShare(share) += add;
    }

    Share result(input.size());
    dReLU(input, result);
    DeviceData<T> super_result(result.size());
    reconstruct(result, super_result);

    std::vector<double> expected = {
        1, 1, 1, 0, 1, 0, 0, 0
    };

    assertShare(super_result, expected, false);
}

TYPED_TEST(GForceTest, DRELU4) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    std::vector<double> input;

    random_vector(input, 20);
    Share a(20);
    a.setPublic(input);

    Share result(input.size());
    dReLU(a, result);

    std::vector<double> expected(20);
    std::transform(input.begin(), input.end(), expected.begin(),
        [](double x) {return x>=0;} );

    assertShare(result, expected, false);
}

TYPED_TEST(GForceTest, DRELU_MAIN) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    std::vector<double> input;

    random_vector(input, 20);
    Share a(20);
    a.setPublic(input);

    DeviceData<T> test_data(20);
    test_data.fill(0);
    test_data += *a.getShare(0);
    test_data += GFORCE_BOUND;
    // printDeviceData(test_data, "plus_bound", false);

    Share result(input.size());
    dReLU(a, result);
    // printDeviceData(*result.getShare(0), "actual", false);

    std::vector<double> expected(20);
    std::transform(input.begin(), input.end(), expected.begin(),
        [](double x) {return x>=0;} );
    DeviceData<T> expt(20);
    thrust::copy(expected.begin(), expected.end(), expt.begin());

    printDeviceData(expt, "expected", false);
    assertShare(result, expected, false);
}





TYPED_TEST(GForceTest, RELU) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    Share input = {
        -2, -3, 4, 3, 3.5, 1, -1.5, -1
    };

    Share result(input.size());
    //Change Share to TPC<uint8_t>
    Share dresult(input.size());
    ReLU(input, result, dresult);

    std::vector<double> expected = {
        0, 0, 4, 3, 3.5, 1, 0, 0
    };

    DeviceData<T> super_result(result.size());
    reconstruct(result, super_result);
    //printDeviceData(super_result, "super_result_64");
    assertDeviceData(super_result, expected);

    std::vector<double> dexpected = {
        0, 0, 1, 1, 1, 1, 0, 0
    };
    
    //Change to <uint8_t>
    reconstruct(dresult, super_result);
    //printDeviceData(super_result, "super_result", false);
    assertDeviceData(super_result, dexpected, false);
}

TYPED_TEST(GForceTest, RELU2) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    Share input = {
        -1.3847, -3, 224.9888, 3.1234567, 3.5, 1.5444332211, -1.511111111, -1
    };

    Share result(input.size());
    //Change Share to TPC<uint8_t>
    Share dresult(input.size());
    ReLU(input, result, dresult);

    std::vector<double> expected = {
       0, 0, 224.9888, 3.1234567, 3.5, 1.5444332211, 0, 0
    };

    DeviceData<T> super_result(result.size());
    reconstruct(result, super_result);
    //printDeviceData(super_result, "super_result_64");
    assertDeviceData(super_result, expected);

    std::vector<double> dexpected = {
        0, 0, 1, 1, 1, 1, 0, 0
    };
    
    //Change to <uint8_t>
    reconstruct(dresult, super_result);
    //printDeviceData(super_result, "super_result", false);
    assertDeviceData(super_result, dexpected, false);
}

TYPED_TEST(GForceTest, Maxpool) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;

    Share input = {1, 3, 4, 3, 7, 1, 2, 10};
    Share result(input.size() / 4);
    Share dresult(input.size());

    maxpool(input, result, dresult, 4);

    std::vector<double> expected = {
        4, 10
    };

    std::vector<double> dexpected = {
        0, 0, 1, 0, 0, 0, 0, 1
    };

    DeviceData<T> super_result(expected.size());
    DeviceData<T> d_super_result(dexpected.size());
    reconstruct(result, super_result);
    assertDeviceData(super_result, expected);

    //Change to <uint8_t>
    reconstruct(dresult, d_super_result);
    //printDeviceData(super_result, "super_result", false);
    assertDeviceData(d_super_result, dexpected, false);
}

TYPED_TEST(GForceTest, Maxpool2) {

    using Share = typename TestFixture::ParamType;
    using T = typename Share::share_type;

    if (partyNum >= Share::numParties) return;
    
    Share input = {-0.032290, -0.142006, -0.031253, 0.130512, -0.301328, -0.105484, 0.002150, 0.055205, 0.234268};
    Share result(1);
    Share dresult(input.size());

    int expandedPoolSize = 16;

   	Share pools((size_t)0);
   	for(int share = 0; share < Share::numShares(); share++) {
	   	gpu::maxpool_im2row(
                input.getShare(share),
                pools.getShare(share),
	   			3, 3, 3, 1, 1,
	   			1, 0,
                -10
                // TODO std::numeric_limits<S>::min() / 3
        );
   	}

    Share expandedMaxPrime(pools.size());
    maxpool(pools, result, expandedMaxPrime, expandedPoolSize);

    // truncate dresult from expanded -> original pool size
    for (int share = 0; share < Share::numShares(); share++) {
        gpu::truncate_cols(expandedMaxPrime.getShare(share), dresult.getShare(share), pools.size() / expandedPoolSize, expandedPoolSize, 9);
    }

    std::vector<double> expected = {
        0.234268
    };

    std::vector<double> dexpected = {
        0, 0, 0, 0, 0, 0, 0, 0, 1
    };
    
    DeviceData<T> super_result(expected.size());
    DeviceData<T> d_super_result(dexpected.size());
    reconstruct(result, super_result);
    assertDeviceData(super_result, expected);

    //Change to <uint8_t>
    reconstruct(dresult, d_super_result);
    //printDeviceData(super_result, "super_result", false);
    assertDeviceData(d_super_result, dexpected, false);
}